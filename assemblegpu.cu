#include "hip/hip_runtime.h"
#include "include/global.hpp"
#include "include/global.cuh"
#include <cassert>

__device__ void add_local_atomic(const size_t* const rowptr, const size_t* const colind, float* const values, const size_t row, const size_t col, const float val)
{
    size_t pos_to_insert(rowptr[row]);
    while (colind[pos_to_insert] < col && pos_to_insert < rowptr[row+1])
        ++pos_to_insert;
    atomicAdd(values + pos_to_insert, val);
}

__global__ void atomic(const size_t* const rowptr, const size_t* const colind, float* const values, const size_t numrows, const FullTriangle* const elements, const size_t numelem)
{
    size_t pos_of_elem(blockDim.x*blockIdx.x + threadIdx.x);
    if (pos_of_elem < numelem)
    {
        FullTriangle elem(elements[pos_of_elem]);
        float B[2][2];
        B[0][0] = elem.nodeB.x - elem.nodeA.x;
        B[1][0] = elem.nodeB.y - elem.nodeA.y;
        B[0][1] = elem.nodeC.x - elem.nodeA.x;
        B[1][1] = elem.nodeC.y - elem.nodeA.y;
        //float d[2];
        //d[0] = elem.nodeA.x;
        //d[1] = elem.nodeA.y;
        float detB(std::abs(B[0][0]*B[1][1] - B[0][1]*B[1][0]));
        float gradA[2], gradB[2], gradC[2]; // in fact these are the gradient multplied by det(B)
        gradA[0] = B[1][0] - B[1][1];
        gradA[1] = B[0][1] - B[0][0];
        gradB[0] = B[1][1];
        gradB[1] = -B[0][1];
        gradC[0] = -B[1][0];
        gradC[1] = B[0][0];

        add_local_atomic(rowptr, colind, values, elem.nodeA.ID, elem.nodeA.ID, (gradA[0]*gradA[0] + gradA[1]*gradA[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeA.ID, elem.nodeB.ID, (gradA[0]*gradB[0] + gradA[1]*gradB[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeA.ID, elem.nodeC.ID, (gradA[0]*gradC[0] + gradA[1]*gradC[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeB.ID, elem.nodeA.ID, (gradB[0]*gradA[0] + gradB[1]*gradA[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeB.ID, elem.nodeB.ID, (gradB[0]*gradB[0] + gradB[1]*gradB[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeB.ID, elem.nodeC.ID, (gradB[0]*gradC[0] + gradB[1]*gradC[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeC.ID, elem.nodeA.ID, (gradC[0]*gradA[0] + gradC[1]*gradA[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeC.ID, elem.nodeB.ID, (gradC[0]*gradB[0] + gradC[1]*gradB[1]) / 2.0 / detB);
        add_local_atomic(rowptr, colind, values, elem.nodeC.ID, elem.nodeC.ID, (gradC[0]*gradC[0] + gradC[1]*gradC[1]) / 2.0 / detB);
    }
}

__global__ void boundary(const size_t* const rowptr, const size_t* const colind, float* const values, const size_t numrows, const size_t* const boundaryNodes, const size_t numboundaryNodes)
{
    size_t id(blockDim.x*blockIdx.x + threadIdx.x);
    if (id < numboundaryNodes)
    {
        size_t b_id{boundaryNodes[id]};
        for (size_t i{rowptr[b_id]}; i < rowptr[b_id+1]; ++i)
            values[i] = (colind[i] == b_id ? 1.0 : 0.0);
    }
}

void assemble_atomic(size_t* d_rowptr, size_t* d_colind, float* d_values, size_t numrows, FullTriangle* h_elements, size_t numelem, size_t* h_boundaryNodes, size_t numboundaryNodes)
{
    FullTriangle* d_elements;
    malloc_cuda(&d_elements, numelem*sizeof(FullTriangle));
    memcpy_cuda(d_elements, h_elements, numelem*sizeof(FullTriangle), h2d);

    dim3 numthreads;
    dim3 numblocks;
    get_kernel_config(&numblocks, &numthreads, numelem);
    atomic<<<numblocks, numthreads>>>(d_rowptr, d_colind, d_values, numrows, d_elements, numelem);
    get_kernel_config(&numblocks, &numthreads, numboundaryNodes);
    hipDeviceSynchronize();
    free_cuda(d_elements);

    size_t* d_boundaryNodes;
    malloc_cuda(&d_boundaryNodes, numboundaryNodes*sizeof(size_t));
    memcpy_cuda(d_boundaryNodes, h_boundaryNodes, numboundaryNodes*sizeof(size_t), h2d);
    boundary<<<numblocks, numthreads>>>(d_rowptr, d_colind, d_values, numrows, d_boundaryNodes, numboundaryNodes);
    hipDeviceSynchronize();

    free_cuda(d_elements);
}
