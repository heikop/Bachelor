#include "include/global.hpp"

template<typename scalar>
void malloc_cuda(scalar** devPtr, size_t size)
{
    hipMalloc(devPtr, size);
}
template void malloc_cuda<size_t>(size_t**, size_t);
template void malloc_cuda<float>(float**, size_t);
template void malloc_cuda<double>(double**, size_t);
template void malloc_cuda<FullTriangle>(FullTriangle**, size_t);

template<typename scalar>
void free_cuda(scalar* devPtr)
{
    hipFree(devPtr);
}
template void free_cuda<size_t>(size_t*);
template void free_cuda<float>(float*);
template void free_cuda<double>(double*);
template void free_cuda<FullTriangle>(FullTriangle*);

void memcpy_cuda(void* dst, const void* src, size_t count, memcpyKind kind)
{
    switch(kind)
    {
    case h2d: hipMemcpy(dst, src, count, hipMemcpyHostToDevice); break;
    case d2h: hipMemcpy(dst, src, count, hipMemcpyDeviceToHost); break;
    //default: //error
    }
}
