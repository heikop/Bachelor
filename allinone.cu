#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "include/csrmatrixgpu.hpp"
#include "include/vectorgpu.hpp"
#include "include/global.cuh"
#include <iostream>

__device__ void add_atomic(const size_t* const rowptr, const size_t* const colind, float* const values, const size_t row, const size_t col, const float val)
{
    size_t pos_to_insert(rowptr[row]);
    while (colind[pos_to_insert] < col && pos_to_insert < rowptr[row+1])
        ++pos_to_insert;
    atomicAdd(values + pos_to_insert, val);
    //*(values + pos_to_insert) += val; // this is not faster !?
}

// needs 18 * sizeo(double) Bytes shared memory per block
// only utilizes x-Dimension
__global__ void assemble_element_Q1_O5(const float* node_coords, const size_t* dof_ids, const float* quadpoints, const float* weights, const size_t num_elements,
                                       const size_t* const rowptr, const size_t* const colind, float* values, const size_t numrows)
{
size_t globalID = blockDim.x * blockIdx.x + threadIdx.x;
if (globalID < num_elements)
{
    const float* coords = node_coords + 8*globalID;
    const size_t* dofs = dof_ids + 4*globalID;
    // load quadrature points and there corresponding weights into shared memory
    __shared__ float sharedmemory[27];
    float* s_quadpoints = sharedmemory;
    float* s_weights    = sharedmemory+18;
    if (threadIdx.x < 27)
    {
        if (threadIdx.x < 18)
            s_quadpoints[threadIdx.x] = quadpoints[threadIdx.x];
        else
            s_weights[threadIdx.x-18] = weights[threadIdx.x-18];
    }

    for (short i(0); i < 4; ++i)
    {
        for (short j(0); j < 4; ++j)
        {
            float val(0.0);

            for (size_t p(0); p < 9; ++p)
            {
                float xi  = s_quadpoints[2*p  ];
                float eta = s_quadpoints[2*p+1];
                float B[2][2] =
                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
                //    { { ( -(1.0f-eta)*node_coords[8*globalID + 0] + (1.0f-eta)*node_coords[8*globalID + 1] + (1.0f+eta)*node_coords[8*globalID + 2] - (1.0f+eta)*node_coords[8*globalID + 3] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 0] - (1.0f+xi )*node_coords[8*globalID + 1] + (1.0f+xi )*node_coords[8*globalID + 2] + (1.0f-xi )*node_coords[8*globalID + 3] ) * 0.25f },
                //      { ( -(1.0f-eta)*node_coords[8*globalID + 4] + (1.0f-eta)*node_coords[8*globalID + 5] + (1.0f+eta)*node_coords[8*globalID + 6] - (1.0f+eta)*node_coords[8*globalID + 7] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 4] - (1.0f+xi )*node_coords[8*globalID + 5] + (1.0f+xi )*node_coords[8*globalID + 6] + (1.0f-xi )*node_coords[8*globalID + 7] ) * 0.25f } };

                // help vars
                float grad1[2];
                float grad2[2];
                grad1[0] = (i == 0 ? (1.0f - eta) * (-0.25f) :
                           (i == 1 ? (1.0f - eta) *   0.25f  :
                           (i == 2 ? (1.0f + eta) *   0.25f  :
                                     (1.0f + eta) * (-0.25f) ) ) );
                grad1[1] = (i == 0 ? (1.0f - xi ) * (-0.25f) :
                           (i == 1 ? (1.0f + xi ) * (-0.25f) :
                           (i == 2 ? (1.0f + xi ) *   0.25f  :
                                     (1.0f - xi ) *   0.25f  ) ) );
                grad2[0] = (j == 0 ? (1.0f - eta) * (-0.25f) :
                           (j == 1 ? (1.0f - eta) *   0.25f  :
                           (j == 2 ? (1.0f + eta) *   0.25f  :
                                     (1.0f + eta) * (-0.25f) ) ) );
                grad2[1] = (j == 0 ? (1.0f - xi ) * (-0.25f) :
                           (j == 1 ? (1.0f + xi ) * (-0.25f) :
                           (j == 2 ? (1.0f + xi ) *   0.25f  :
                                     (1.0f - xi ) *   0.25f  ) ) );

                val  += s_weights[p]
                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
            } // end for p (quadrature point)
            add_atomic(rowptr, colind, values, dofs[i], dofs[j], val);
        } // end for j
    } // end for i
}// end if globalID < num_elements
}

void __K_assemble_element_Q1_O5(const float* d_coords, const size_t* d_dof, const float* d_quadpoints, const float* d_weights, const size_t num_elements,
                                const size_t* const d_rowptr, const size_t* const d_colind, float* d_values, const size_t numrows)
{
    dim3 numthreads;
    dim3 numblocks;
    get_kernel_config(&numblocks, &numthreads, num_elements);
    assemble_element_Q1_O5<<<numblocks, numthreads>>>(d_coords, d_dof, d_quadpoints, d_weights, num_elements,
                                                      d_rowptr, d_colind, d_values, numrows);
    hipDeviceSynchronize();
}



// ***** ***** ***** LOCAL MATRIX APPROACH ***** ***** *****

__global__ void assemble_element_Q1_O5_LM(const float* node_coords, const size_t* dof_ids, const float* quadpoints, const float* weights, const size_t num_elements,
                                          float* localmatrices)
{
size_t globalID = blockDim.x * blockIdx.x + threadIdx.x;
if (globalID < num_elements)
{
    const float* coords = node_coords + 8*globalID;
    //const size_t* dofs = dof_ids + 4*globalID;
    float* LM = localmatrices + 16*globalID;
    // load quadrature points and there corresponding weights into shared memory
    __shared__ float sharedmemory[27];
    float* s_quadpoints = sharedmemory;
    float* s_weights    = sharedmemory+18;
    if (threadIdx.x < 27)
    {
        if (threadIdx.x < 18)
            s_quadpoints[threadIdx.x] = quadpoints[threadIdx.x];
        else
            s_weights[threadIdx.x-18] = weights[threadIdx.x-18];
    }

    for (short i(0); i < 4; ++i)
    {
        for (short j(0); j < 4; ++j)
        {
            float val(0.0);

            for (size_t p(0); p < 9; ++p)
            {
                float xi  = s_quadpoints[2*p  ];
                float eta = s_quadpoints[2*p+1];
                float B[2][2] =
                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
                //    { { ( -(1.0f-eta)*node_coords[8*globalID + 0] + (1.0f-eta)*node_coords[8*globalID + 1] + (1.0f+eta)*node_coords[8*globalID + 2] - (1.0f+eta)*node_coords[8*globalID + 3] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 0] - (1.0f+xi )*node_coords[8*globalID + 1] + (1.0f+xi )*node_coords[8*globalID + 2] + (1.0f-xi )*node_coords[8*globalID + 3] ) * 0.25f },
                //      { ( -(1.0f-eta)*node_coords[8*globalID + 4] + (1.0f-eta)*node_coords[8*globalID + 5] + (1.0f+eta)*node_coords[8*globalID + 6] - (1.0f+eta)*node_coords[8*globalID + 7] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 4] - (1.0f+xi )*node_coords[8*globalID + 5] + (1.0f+xi )*node_coords[8*globalID + 6] + (1.0f-xi )*node_coords[8*globalID + 7] ) * 0.25f } };

                // help vars
                float grad1[2];
                float grad2[2];
                grad1[0] = (i == 0 ? (1.0f - eta) * (-0.25f) :
                           (i == 1 ? (1.0f - eta) *   0.25f  :
                           (i == 2 ? (1.0f + eta) *   0.25f  :
                                     (1.0f + eta) * (-0.25f) ) ) );
                grad1[1] = (i == 0 ? (1.0f - xi ) * (-0.25f) :
                           (i == 1 ? (1.0f + xi ) * (-0.25f) :
                           (i == 2 ? (1.0f + xi ) *   0.25f  :
                                     (1.0f - xi ) *   0.25f  ) ) );
                grad2[0] = (j == 0 ? (1.0f - eta) * (-0.25f) :
                           (j == 1 ? (1.0f - eta) *   0.25f  :
                           (j == 2 ? (1.0f + eta) *   0.25f  :
                                     (1.0f + eta) * (-0.25f) ) ) );
                grad2[1] = (j == 0 ? (1.0f - xi ) * (-0.25f) :
                           (j == 1 ? (1.0f + xi ) * (-0.25f) :
                           (j == 2 ? (1.0f + xi ) *   0.25f  :
                                     (1.0f - xi ) *   0.25f  ) ) );

                val  += s_weights[p]
                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
            } // end for p (quadrature point)
            //add_atomic(rowptr, colind, values, dof_ids[4*globalID + i], dof_ids[4*globalID + j], val);
            //add_atomic(rowptr, colind, values, dofs[i], dofs[j], val);
            LM[4*i + j] = val;
        } // end for j
    } // end for i
}// end if globalID < num_elements
}


//__global__ void assemble_element_Q1_O5_LM(const float* node_coords, const size_t* dof_ids, const float* quadpoints, const float* weights, const size_t num_elements,
//                                          float* localmatrices)
//{
//size_t globalID = blockDim.x * blockIdx.x + threadIdx.x;
//if (globalID < num_elements)
//{
//    const float* coords = node_coords + 8*globalID;
//    //const size_t* dofs = dof_ids + 4*globalID;
//    float* LM = localmatrices + 16*globalID;
//    // load quadrature points and there corresponding weights into shared memory
//    __shared__ float sharedmemory[27];
//    float* s_quadpoints = sharedmemory;
//    float* s_weights    = sharedmemory+18;
//    if (threadIdx.x < 27)
//    {
//        if (threadIdx.x < 18)
//            s_quadpoints[threadIdx.x] = quadpoints[threadIdx.x];
//        else
//            s_weights[threadIdx.x-18] = weights[threadIdx.x-18];
//    }
//
//    // i = 0
//        for (short j(0); j < 4; ++j)
//        {
//            float val(0.0);
//
//            for (size_t p(0); p < 9; ++p)
//            {
//                float xi  = s_quadpoints[2*p  ];
//                float eta = s_quadpoints[2*p+1];
//                float B[2][2] =
//                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
//                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
//
//                // help vars
//                float grad1[2];
//                float grad2[2];
//                grad1[0] = (1.0f - eta) * (-0.25f);
//                grad1[1] = (1.0f - xi ) * (-0.25f);
//                grad2[0] = (1.0f - eta) * (-0.25f);
//                grad2[1] = (1.0f - xi ) * (-0.25f);
//
//                val  += s_weights[p]
//                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
//                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
//                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
//            } // end for p (quadrature point)
//            LM[j] = val;
//        } // end for j
//    // i = 1
//        for (short j(0); j < 4; ++j)
//        {
//            float val(0.0);
//
//            for (size_t p(0); p < 9; ++p)
//            {
//                float xi  = s_quadpoints[2*p  ];
//                float eta = s_quadpoints[2*p+1];
//                float B[2][2] =
//                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
//                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
//
//                // help vars
//                float grad1[2];
//                float grad2[2];
//                grad1[0] = (1.0f - eta) *   0.25f ;
//                grad1[1] = (1.0f + xi ) * (-0.25f);
//                grad2[0] = (1.0f - eta) *   0.25f ;
//                grad2[1] = (1.0f + xi ) * (-0.25f);
//
//                val  += s_weights[p]
//                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
//                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
//                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
//            } // end for p (quadrature point)
//            LM[4 + j] = val;
//        } // end for j
//    // i = 2
//        for (short j(0); j < 4; ++j)
//        {
//            float val(0.0);
//
//            for (size_t p(0); p < 9; ++p)
//            {
//                float xi  = s_quadpoints[2*p  ];
//                float eta = s_quadpoints[2*p+1];
//                float B[2][2] =
//                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
//                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
//
//                // help vars
//                float grad1[2];
//                float grad2[2];
//                grad1[0] = (1.0f + eta) *   0.25f ;
//                grad1[1] = (1.0f + xi ) *   0.25f ;
//                grad2[0] = (1.0f + eta) *   0.25f ;
//                grad2[1] = (1.0f + xi ) *   0.25f ;
//
//                val  += s_weights[p]
//                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
//                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
//                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
//            } // end for p (quadrature point)
//            LM[8 + j] = val;
//        } // end for j
//    // i = 3
//        for (short j(0); j < 4; ++j)
//        {
//            float val(0.0);
//
//            for (size_t p(0); p < 9; ++p)
//            {
//                float xi  = s_quadpoints[2*p  ];
//                float eta = s_quadpoints[2*p+1];
//                float B[2][2] =
//                    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
//                      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
//                        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
//
//                // help vars
//                float grad1[2];
//                float grad2[2];
//                grad1[0] = (1.0f + eta) * (-0.25f);
//                grad1[1] = (1.0f - xi ) *   0.25f ;
//                grad2[0] = (1.0f + eta) * (-0.25f);
//                grad2[1] = (1.0f - xi ) *   0.25f ;
//
//                val  += s_weights[p]
//                        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
//                            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
//                        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
//            } // end for p (quadrature point)
//            LM[12 + j] = val;
//        } // end for j
//
//}// end if globalID < num_elements
//}

void gather_res(const size_t* d_dof, const float* const d_localmatrices, const size_t numelem)//, float* res)
{
    //size_t h_dof[4*numelem];
    size_t h_dof[4];
    //memcpy_cuda(h_dof, d_dof, 4*numelem*sizeof(size_t), d2h);
    for (size_t e(0); e < numelem; ++e)
    {
        memcpy_cuda(h_dof, d_dof+4*e, 4*sizeof(size_t), d2h);
        for (size_t k(0); k < 4; ++k)
            if (h_dof[k] == 0)
            {
                float tmp[4];
                memcpy_cuda(tmp, d_localmatrices + 16*e + 4*k, 4*sizeof(float), d2h);
                std::cout << h_dof[0] << ": " << tmp[0] << std::endl;
                std::cout << h_dof[1] << ": " << tmp[1] << std::endl;
                std::cout << h_dof[2] << ": " << tmp[2] << std::endl;
                std::cout << h_dof[3] << ": " << tmp[3] << std::endl;
            }
    }
}


void __K_assemble_element_Q1_O5_LM(const float* d_coords, const size_t* d_dof, const float* d_quadpoints, const float* d_weights, const size_t num_elements,
                                   float* d_localmatrices)
{
    dim3 numthreads;
    dim3 numblocks;
    get_kernel_config(&numblocks, &numthreads, num_elements);
    std::cout << numblocks.x << " blocks a " << numthreads.x << " threads" << std::endl;
    assemble_element_Q1_O5_LM<<<numblocks, numthreads>>>(d_coords, d_dof, d_quadpoints, d_weights, num_elements,
                                                         d_localmatrices);
    hipDeviceSynchronize();
    //gather_res(d_dof, d_localmatrices, num_elements);
}

// on element per block

//__global__ void assemble_element_Q1_O5_LM_1epb(const float* node_coords, const float* quadpoints,
//                                               const float* weights, float* localmatrices)
__global__ void assemble_element_Q1_O5_LM_1epb(const float* node_coords, const float* quadpoints_weights, float* localmatrices)
{
    const float* coords = node_coords + 8*blockIdx.x;
    //const size_t* dofs = dof_ids + 4*globalID;
    float* LM = localmatrices + 16*blockIdx.x;
    // load quadrature points and there corresponding weights and the coordinagtes into shared memory
    __shared__ float sharedmemory[35 + 144]; // 2*9qp + 9w + 2*4co + 4*4*9val
    float* s_quadpoints = sharedmemory;
    float* s_weights    = sharedmemory + 18;
    float* s_coords     = sharedmemory + 27;

    int i = threadIdx.x;
    int j = threadIdx.y;
    s_quadpoints[4*i + j] = quadpoints_weights[4*i + j];
    if (4*i + j < 11)
        s_quadpoints[16 + 4*i + j] = quadpoints_weights[16 + 4*i + j];
    if (i < 2)
        s_coords[2*i + j] = coords[2*i + j];
            //float val(0.0);

            //for (size_t p(0); p < 9; ++p)
            //{
            int p = threadIdx.z;
                float xi  = s_quadpoints[2*p  ];
                float eta = s_quadpoints[2*p+1];
                float B[2][2] =
                    { { ( -(1.0f-eta)*s_coords[0] + (1.0f-eta)*s_coords[1] + (1.0f+eta)*s_coords[2] - (1.0f+eta)*s_coords[3] ) * 0.25f ,
                        ( -(1.0f-xi )*s_coords[0] - (1.0f+xi )*s_coords[1] + (1.0f+xi )*s_coords[2] + (1.0f-xi )*s_coords[3] ) * 0.25f },
                      { ( -(1.0f-eta)*s_coords[4] + (1.0f-eta)*s_coords[5] + (1.0f+eta)*s_coords[6] - (1.0f+eta)*s_coords[7] ) * 0.25f ,
                        ( -(1.0f-xi )*s_coords[4] - (1.0f+xi )*s_coords[5] + (1.0f+xi )*s_coords[6] + (1.0f-xi )*s_coords[7] ) * 0.25f } };
                //    { { ( -(1.0f-eta)*coords[0] + (1.0f-eta)*coords[1] + (1.0f+eta)*coords[2] - (1.0f+eta)*coords[3] ) * 0.25f ,
                //        ( -(1.0f-xi )*coords[0] - (1.0f+xi )*coords[1] + (1.0f+xi )*coords[2] + (1.0f-xi )*coords[3] ) * 0.25f },
                //      { ( -(1.0f-eta)*coords[4] + (1.0f-eta)*coords[5] + (1.0f+eta)*coords[6] - (1.0f+eta)*coords[7] ) * 0.25f ,
                //        ( -(1.0f-xi )*coords[4] - (1.0f+xi )*coords[5] + (1.0f+xi )*coords[6] + (1.0f-xi )*coords[7] ) * 0.25f } };
                //    { { ( -(1.0f-eta)*node_coords[8*globalID + 0] + (1.0f-eta)*node_coords[8*globalID + 1] + (1.0f+eta)*node_coords[8*globalID + 2] - (1.0f+eta)*node_coords[8*globalID + 3] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 0] - (1.0f+xi )*node_coords[8*globalID + 1] + (1.0f+xi )*node_coords[8*globalID + 2] + (1.0f-xi )*node_coords[8*globalID + 3] ) * 0.25f },
                //      { ( -(1.0f-eta)*node_coords[8*globalID + 4] + (1.0f-eta)*node_coords[8*globalID + 5] + (1.0f+eta)*node_coords[8*globalID + 6] - (1.0f+eta)*node_coords[8*globalID + 7] ) * 0.25f ,
                //        ( -(1.0f-xi )*node_coords[8*globalID + 4] - (1.0f+xi )*node_coords[8*globalID + 5] + (1.0f+xi )*node_coords[8*globalID + 6] + (1.0f-xi )*node_coords[8*globalID + 7] ) * 0.25f } };

                // help vars
                float grad1[2];
                float grad2[2];

                grad1[0] = (i < 2 ? (1.0f - eta) : (1.0f + eta) );
                grad1[0] *= (i == 0 || i == 3 ? -0.25f : 0.25f );
                grad1[1] = (i < 2 ? -0.25f : 0.25f);
                grad1[1] *= (i == 0 || i == 3 ? (1.0f - xi ) : (1.0f + xi ) );
                grad2[0] = (j < 2 ? (1.0f - eta) : (1.0f + eta) );
                grad2[0] *= (j == 0 || j == 3 ? -0.25f : 0.25f );
                grad2[1] = (j < 2 ? -0.25f : 0.25f);
                grad2[1] *= (j == 0 || j == 3 ? (1.0f - xi ) : (1.0f + xi ) );

                //grad1[0] = (i == 0 ? (1.0f - eta) * (-0.25f) :
                //           (i == 1 ? (1.0f - eta) *   0.25f  :
                //           (i == 2 ? (1.0f + eta) *   0.25f  :
                //                     (1.0f + eta) * (-0.25f) ) ) );
                //grad1[1] = (i == 0 ? (1.0f - xi ) * (-0.25f) :
                //           (i == 1 ? (1.0f + xi ) * (-0.25f) :
                //           (i == 2 ? (1.0f + xi ) *   0.25f  :
                //                     (1.0f - xi ) *   0.25f  ) ) );
                //grad2[0] = (j == 0 ? (1.0f - eta) * (-0.25f) :
                //           (j == 1 ? (1.0f - eta) *   0.25f  :
                //           (j == 2 ? (1.0f + eta) *   0.25f  :
                //                     (1.0f + eta) * (-0.25f) ) ) );
                //grad2[1] = (j == 0 ? (1.0f - xi ) * (-0.25f) :
                //           (j == 1 ? (1.0f + xi ) * (-0.25f) :
                //           (j == 2 ? (1.0f + xi ) *   0.25f  :
                //                     (1.0f - xi ) *   0.25f  ) ) );

                //val  += s_weights[p]
                //        * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
                //            + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
                //        / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
                sharedmemory[35 + 9*(4*i + j) + p] = s_weights[p]
                                                 * (   ( B[1][1] * grad1[0] - B[1][0] * grad1[1]) * ( B[1][1] * grad2[0] - B[1][0] * grad2[1])
                                                     + (-B[0][1] * grad1[0] + B[0][0] * grad1[1]) * (-B[0][1] * grad2[0] + B[0][0] * grad2[1]) )
                                                 / std::abs(B[0][0] * B[1][1] - B[0][1] * B[1][0]);
            //} // end for p (quadrature point)
            __syncthreads();
            if (p == 0)
            {
                float val(0.0f);
                for (int k(0); k < 9; ++k)
                    val += sharedmemory[35 + 4*i + j + p];
                LM[4*i + j] = val;
            }
            //LM[4*i + j] = val;
    //    } // end for j
    //} // end for i
}

void __K_assemble_element_Q1_O5_LM_1epb(const float* d_coords, const float* d_quadpoints, const float* d_weights,
                                        const size_t num_elements, float* d_localmatrices)
{
    //assemble_element_Q1_O5_LM_1epb<<<dim3(num_elements, 1, 1), dim3(4, 4, 1)>>>(d_coords, d_quadpoints, d_weights, d_localmatrices);
    assemble_element_Q1_O5_LM_1epb<<<dim3(num_elements, 1, 1), dim3(4, 4, 9)>>>(d_coords, d_quadpoints, d_localmatrices);
    hipDeviceSynchronize();
}
