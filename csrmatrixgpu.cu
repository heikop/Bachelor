#include "hip/hip_runtime.h"
#include "include/csrmatrixgpu.hpp"
#include "include/global.cuh"

//***** multvec *****//

__global__ void csrmatrix_multvec_kernel(const size_t* const rowptr, const size_t* const colind, const float* const values, const float* const vec, float* const res, size_t numrows)
{
    size_t row(blockDim.x*blockIdx.x + threadIdx.x);
    float tmpres(0.0);
    if (row < numrows)
        for (size_t col(rowptr[row]); col < rowptr[row+1]; ++col)
            tmpres += values[col] * vec[colind[col]];
    res[row] = tmpres;
}

void CsrMatrixGpu::multvec(const VectorGpu& vec, VectorGpu& res) const
{
    assert(_numcols == vec._size && _numrows == res._size); //TODISCUSS or reallocate when res has a different size?
    dim3 numblocks, numthreads;
    get_kernel_config(&numblocks, &numthreads, _numrows);
    csrmatrix_multvec_kernel<<<numblocks, numthreads>>>(_rowptr, _colind, _values, vec._values, res._values, _numrows);
    hipDeviceSynchronize();
}

//***** create structure *****//

__global__ void set_zeros(float* values, size_t num_values)
{
    size_t id(blockDim.x*blockIdx.x + threadIdx.x);
    if (id < num_values)
        values[id] = 0.0;
}

